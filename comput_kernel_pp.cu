#include "hip/hip_runtime.h"
//Add parallel reduction to find mininum impurity based on kernel_2.cu
#include<stdio.h>
#include<math.h>
#define MAX_NUM_SAMPLES %d
#define MAX_NUM_LABELS %d
#define MAX_THREADS_PER_BLOCK 256

__device__  float calc_imp_right(int label_previous[MAX_NUM_LABELS], int label_now[MAX_NUM_LABELS], int total_size){
  float imp = 1.0;
  for(int i = 0; i < MAX_NUM_LABELS; ++i)
    imp -= pow(((label_now[i] - label_previous[i]) / double(total_size)), 2); 

  return imp; 
}

__device__  float calc_imp_left(int label_now[MAX_NUM_LABELS], int total_size){
  float imp = 1.0;
  for(int i = 0; i < MAX_NUM_LABELS; ++i)
    imp -= pow((label_now[i] / double(total_size)), 2); 

  return imp; 
}

__global__ void compute(float *sorted_samples, 
                        float *imp_left, 
                        float *imp_right, 
                        int *label_count,
                        int *split, 
                        int n_features, 
                        int n_samples, 
                        int leading){
  
  int offset = blockIdx.x * MAX_NUM_LABELS * n_samples; 
  __shared__ int quit;
  __shared__ float shared_imp_left[MAX_THREADS_PER_BLOCK];
  __shared__ float shared_imp_right[MAX_THREADS_PER_BLOCK];
  __shared__ int shared_split_index[MAX_THREADS_PER_BLOCK];

  int range = ceil(double(n_samples) / blockDim.x);
  int range_begin =(threadIdx.x * range < n_samples)? threadIdx.x * range : n_samples - 1;
  int range_end = (range_begin + range < n_samples)? range_begin + range : n_samples - 1;
  shared_imp_left[threadIdx.x] = 2;
  shared_imp_right[threadIdx.x] = 2;

  if(threadIdx.x == 0){
    if(sorted_samples[blockIdx.x * n_samples] == sorted_samples[blockIdx.x * n_samples + n_samples - 1]){
      imp_left[blockIdx.x] = 2;
      imp_right[blockIdx.x] = 2;
      quit = 1;
    }
    else
      quit = 0;
  }

  __syncthreads();

  if(quit == 1)
    return;

  for(int i = range_begin; i < range_end; ++i){
    float cur_value = sorted_samples[blockIdx.x * n_samples + i];
    float next_value = sorted_samples[blockIdx.x * n_samples + i + 1];
    if(cur_value == next_value)
      continue;

    float imp_left = ((i + 1) / float(n_samples)) * calc_imp_left(&label_count[i * MAX_NUM_LABELS + offset], i + 1);
    float imp_right = ((n_samples - i - 1) / float(n_samples)) * calc_imp_right(&label_count[i * MAX_NUM_LABELS + offset], &label_count[MAX_NUM_LABELS * (n_samples-1) + offset], n_samples - i - 1);
    float impurity = imp_left + imp_right;
    if(impurity < shared_imp_left[threadIdx.x] + shared_imp_right[threadIdx.x]){
      shared_imp_left[threadIdx.x] = imp_left;
      shared_imp_right[threadIdx.x] = imp_right;
      shared_split_index[threadIdx.x] = i;
    }
  }   
  __syncthreads();

  int n_threads = blockDim.x;
  int next_thread;

  //Parallel tree reduction to find mininum impurity
  while(n_threads > 1){
    int half = (n_threads >> 1);
    if(threadIdx.x < half){
      next_thread = threadIdx.x + half;
      if(shared_imp_left[threadIdx.x] + shared_imp_right[threadIdx.x] > shared_imp_left[next_thread] + shared_imp_right[next_thread]){
        shared_imp_left[threadIdx.x] = shared_imp_left[next_thread];
        shared_imp_right[threadIdx.x] = shared_imp_right[next_thread];
        shared_split_index[threadIdx.x] = shared_split_index[next_thread];
      }
    }
    
    n_threads = half;
  }

  __syncthreads();

  if(threadIdx.x != 0)
    return;

  imp_left[blockIdx.x] = shared_imp_left[0];
  imp_right[blockIdx.x] = shared_imp_right[0];
  split[blockIdx.x] = shared_split_index[0]; 
}
