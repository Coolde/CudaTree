#include "hip/hip_runtime.h"
//Add parallel reduction to find mininum impurity based on kernel_2.cu
#include<stdio.h>
#include<math.h>
#include<stdint.h>
#define MAX_NUM_SAMPLES %d
#define MAX_NUM_LABELS %d
#define MAX_THREADS_PER_BLOCK 256 
#define SAMPLE_DATA_TYPE %s

__device__  float calc_imp_right(int label_previous[MAX_NUM_LABELS], int label_now[MAX_NUM_LABELS], int total_size){
  float sum = 0.0; 
  for(int i = 0; i < MAX_NUM_LABELS; ++i){
    float count = label_now[i] - label_previous[i];
    sum += count * count;
  }

  float denom = ((float) total_size) * total_size;

  return 1.0 - (sum / denom); 
}

__device__  float calc_imp_left(int label_now[MAX_NUM_LABELS], int total_size){
  float sum = 0.0;
  for(int i = 0; i < MAX_NUM_LABELS; ++i){
    float count = label_now[i];
    sum += count * count;
  }
  
  float denom = ((float) total_size) * total_size;
  return 1.0 - (sum / denom); 
}

__global__ void compute(SAMPLE_DATA_TYPE *sorted_samples, 
                        int *sorted_labels,
                        float *imp_left, 
                        float *imp_right, 
                        int *label_count,
                        int *split, 
                        int n_features, 
                        int n_samples, 
                        int stride){

  int count_offset = blockIdx.x * MAX_NUM_LABELS * (blockDim.x + 1); 
  int samples_offset = blockIdx.x * stride;
  int labels_offset = blockIdx.x * stride;
  
  __shared__ int quit;
  __shared__ float shared_imp_left[MAX_THREADS_PER_BLOCK];
  __shared__ float shared_imp_right[MAX_THREADS_PER_BLOCK];
  __shared__ int shared_split_index[MAX_THREADS_PER_BLOCK];

  int range = ceil(double(n_samples) / blockDim.x);
  int n_active_threads = ceil(double(n_samples) / range);     //The number of threads that have the actual work to do.
  int range_begin =(threadIdx.x * range < n_samples)? threadIdx.x * range : n_samples - 1;
  int range_end = (range_begin + range < n_samples)? range_begin + range : n_samples - 1;
  
  shared_imp_left[threadIdx.x] = 2;
  shared_imp_right[threadIdx.x] = 2;

  if(threadIdx.x == 0){
    if(sorted_samples[samples_offset] == sorted_samples[samples_offset + n_samples - 1]){
      imp_left[blockIdx.x] = 2;
      imp_right[blockIdx.x] = 2; 
      quit = 1;
    }
    else
      quit = 0;
  }

  __syncthreads();

  if(quit == 1)
    return; 

  for(int i = range_begin; i < range_end; ++i){
    int label_val = sorted_labels[labels_offset + i];
    label_count[count_offset + threadIdx.x * MAX_NUM_LABELS + label_val]++;
    
    if(sorted_samples[samples_offset + i] == sorted_samples[samples_offset + i + 1])
      continue;
    
    float imp_left = ((i + 1) / float(n_samples)) * calc_imp_left(&label_count[count_offset + threadIdx.x * MAX_NUM_LABELS], i + 1);
    float imp_right = ((n_samples - i - 1) / float(n_samples)) * calc_imp_right(&label_count[count_offset + threadIdx.x * MAX_NUM_LABELS],
                                                                                &label_count[count_offset + n_active_threads * MAX_NUM_LABELS], n_samples - i - 1);
    
    float impurity = imp_left + imp_right;
    if(impurity < shared_imp_left[threadIdx.x] + shared_imp_right[threadIdx.x]){
      shared_imp_left[threadIdx.x] = imp_left;
      shared_imp_right[threadIdx.x] = imp_right;
      shared_split_index[threadIdx.x] = i;
    }  
  }
  
  __syncthreads();
 
  int n_threads = blockDim.x;
  int next_thread;

  //Parallel tree reduction to find mininum impurity
  while(n_threads > 1){
    int half = (n_threads >> 1);
    if(threadIdx.x < half){
      next_thread = threadIdx.x + half;
      if(shared_imp_left[threadIdx.x] + shared_imp_right[threadIdx.x] > shared_imp_left[next_thread] + shared_imp_right[next_thread]){
        shared_imp_left[threadIdx.x] = shared_imp_left[next_thread];
        shared_imp_right[threadIdx.x] = shared_imp_right[next_thread];
        shared_split_index[threadIdx.x] = shared_split_index[next_thread];
      }
    }

    __syncthreads(); 
    n_threads = half;
  }
 
  __syncthreads();
  imp_left[blockIdx.x] = shared_imp_left[0];
  imp_right[blockIdx.x] = shared_imp_right[0];
  split[blockIdx.x] = shared_split_index[0];  
}
