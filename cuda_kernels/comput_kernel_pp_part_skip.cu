#include "hip/hip_runtime.h"
//Add parallel reduction to find mininum impurity based on kernel_2.cu
#include<stdio.h>
#include<math.h>
#include<stdint.h>
#define MAX_NUM_SAMPLES %d
#define MAX_NUM_LABELS %d
#define MAX_THREADS_PER_BLOCK 256 
#define SAMPLE_DATA_TYPE %s

__device__  float calc_imp_right(int label_previous[MAX_NUM_LABELS], int label_now[MAX_NUM_LABELS], int total_size){
  float sum = 0.0; 
  for(int i = 0; i < MAX_NUM_LABELS; ++i){
    float count = label_now[i] - label_previous[i];
    sum += count * count;
  }

  float denom = ((float) total_size) * total_size;

  return 1.0 - (sum / denom); 
}

__device__  float calc_imp_left(int label_now[MAX_NUM_LABELS], int total_size){
  float sum = 0.0;
  for(int i = 0; i < MAX_NUM_LABELS; ++i){
    float count = label_now[i];
    sum += count * count;
  }
  
  float denom = ((float) total_size) * total_size;
  return 1.0 - (sum / denom); 
}


__device__ int skip(float imp_cur, float imp_min, int n_samples, int n_left){
  if(imp_cur < imp_min){
    printf("!!!!!!!!!!!!###############!!!!!!!!!!!\n");
    return 0;
  }
  
  float dScore = imp_min - imp_cur;
  return floor(-n_samples * dScore / 2 - n_left + sqrt(n_samples * n_samples * dScore * dScore + 4 * n_left * n_left)/2); 
}

__global__ void compute(SAMPLE_DATA_TYPE *sorted_samples, 
                        int *sorted_labels,
                        float *imp_left, 
                        float *imp_right, 
                        int *label_count,
                        int *split, 
                        int n_features, 
                        int n_samples, 
                        int stride){
  int count_offset = blockIdx.x * MAX_NUM_LABELS * (blockDim.x + 1); 
  int samples_offset = blockIdx.x * stride;
  int labels_offset = blockIdx.x * stride;
  
  __shared__ int quit;
  __shared__ float shared_imp_left[MAX_THREADS_PER_BLOCK];
  __shared__ float shared_imp_right[MAX_THREADS_PER_BLOCK];
  __shared__ int shared_split_index[MAX_THREADS_PER_BLOCK];
  __shared__ float shared_min_imp;

  int range = ceil(double(n_samples) / blockDim.x);
  int n_active_threads = ceil(double(n_samples) / range);     //The number of threads that have the actual work to do.
  int range_begin =(threadIdx.x * range < n_samples)? threadIdx.x * range : n_samples - 1;
  int range_end = (range_begin + range < n_samples)? range_begin + range : n_samples - 1;
  
  shared_imp_left[threadIdx.x] = 2;
  shared_imp_right[threadIdx.x] = 2;

  if(threadIdx.x == 0){
    shared_min_imp = 4.0;

    if(sorted_samples[samples_offset] == sorted_samples[samples_offset + n_samples - 1]){
      imp_left[blockIdx.x] = 2;
      imp_right[blockIdx.x] = 2; 
      quit = 1;
    }
    else
      quit = 0;
  }

  __syncthreads();
  

  if(quit == 1)
    return; 
   
  int pos = range_begin;
  for(; pos < range_end; ++pos){
    int label_val = sorted_labels[labels_offset + pos];
    label_count[count_offset + threadIdx.x * MAX_NUM_LABELS + label_val]++;
    
    if(sorted_samples[samples_offset + pos] == sorted_samples[samples_offset + pos + 1])
      continue;
     
    float imp_left = ((pos + 1) / float(n_samples)) * calc_imp_left(&label_count[count_offset + threadIdx.x * MAX_NUM_LABELS], pos + 1);
    float imp_right = ((n_samples - pos - 1) / float(n_samples)) * calc_imp_right(&label_count[count_offset + threadIdx.x * MAX_NUM_LABELS],
                                                                                  &label_count[count_offset + n_active_threads * MAX_NUM_LABELS], n_samples - pos - 1); 
    
    shared_imp_left[threadIdx.x] = imp_left;
    shared_imp_right[threadIdx.x] = imp_right;
    shared_split_index[threadIdx.x] = pos;
    pos++;
    break;
  }
  __syncthreads();

  float imp_l = shared_imp_left[threadIdx.x];
  float imp_r = shared_imp_right[threadIdx.x];
  
  int n_threads = blockDim.x;
  int next_thread;

  while(n_threads > 1){
    int half = (n_threads >> 1);
    if(threadIdx.x < half){
      next_thread = threadIdx.x + half;
      if(shared_imp_left[threadIdx.x] + shared_imp_right[threadIdx.x] > shared_imp_left[next_thread] + shared_imp_right[next_thread]){
        shared_imp_left[threadIdx.x] = shared_imp_left[next_thread];
        shared_imp_right[threadIdx.x] = shared_imp_right[next_thread];
        shared_split_index[threadIdx.x] = shared_split_index[next_thread];
      }
    }
    
    __syncthreads(); 
    n_threads = half;
  }
 
  if(threadIdx.x == 0)
    shared_min_imp = shared_imp_left[0] + shared_imp_right[0];

  shared_imp_left[threadIdx.x] = imp_l;
  shared_imp_right[threadIdx.x] = imp_r;
  
  __syncthreads(); 
  
  bool first = true;

  for(int i = pos; i < range_end; ++i){
    int label_val = sorted_labels[labels_offset + i];
    label_count[count_offset + threadIdx.x * MAX_NUM_LABELS + label_val]++;
    
    if(sorted_samples[samples_offset + i] == sorted_samples[samples_offset + i + 1])
      continue;
    
    float imp_left = ((i + 1) / float(n_samples)) * calc_imp_left(&label_count[count_offset + threadIdx.x * MAX_NUM_LABELS], i + 1);
    float imp_right = ((n_samples - i - 1) / float(n_samples)) * calc_imp_right(&label_count[count_offset + threadIdx.x * MAX_NUM_LABELS],
                                                                                &label_count[count_offset + n_active_threads * MAX_NUM_LABELS], n_samples - i - 1);
    
    float impurity = imp_left + imp_right;
   

    if(impurity < shared_imp_left[threadIdx.x] + shared_imp_right[threadIdx.x]){
      shared_imp_left[threadIdx.x] = imp_left;
      shared_imp_right[threadIdx.x] = imp_right;
      shared_split_index[threadIdx.x] = i;
    }  
    
    if(first)
    {  
      int sk = skip(impurity, shared_min_imp, n_samples, pos);
      //printf("%%f %%f\n", impurity, shared_min_imp);
      
      if(sk > range_end - i)
      { 
        //if(sk > 100)
          printf("skip : %%d\n", sk); 
        break;
      }
      first = false;
    }
  }
  
  __syncthreads();
 
  n_threads = blockDim.x;

  //Parallel tree reduction to find mininum impurity
  while(n_threads > 1){
    int half = (n_threads >> 1);
    if(threadIdx.x < half){
      next_thread = threadIdx.x + half;
      if(shared_imp_left[threadIdx.x] + shared_imp_right[threadIdx.x] > shared_imp_left[next_thread] + shared_imp_right[next_thread]){
        shared_imp_left[threadIdx.x] = shared_imp_left[next_thread];
        shared_imp_right[threadIdx.x] = shared_imp_right[next_thread];
        shared_split_index[threadIdx.x] = shared_split_index[next_thread];
      }
    }

    __syncthreads(); 
    n_threads = half;
  }
 
  __syncthreads();
  
  if(threadIdx.x != 0)
    return;
  
  imp_left[blockIdx.x] = shared_imp_left[0];
  imp_right[blockIdx.x] = shared_imp_right[0];
  split[blockIdx.x] = shared_split_index[0];  
}
