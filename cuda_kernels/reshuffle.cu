#include "hip/hip_runtime.h"
#include<stdio.h>
#include<math.h>
#include<stdint.h>
#define SAMPLE_DATA_TYPE %s
#define LABEL_DATA_TYPE %s
#define IDX_DATA_TYPE %s


__global__ void reshuffle(IDX_DATA_TYPE* mark_table,
                          LABEL_DATA_TYPE* sorted_labels,
                          IDX_DATA_TYPE* sorted_indices,
                          SAMPLE_DATA_TYPE* sorted_samples,
                          LABEL_DATA_TYPE* sorted_labels_out,
                          IDX_DATA_TYPE* sorted_indices_out,
                          SAMPLE_DATA_TYPE* sorted_samples_out,
                          int n_samples,
                          int split_idx,
                          int stride
                          ){
  int offset = blockIdx.x * stride;
  int left_start = 0;
  int right_start = split_idx + 1;

  for(int i = 0; i < n_samples; ++i){
    if(mark_table[sorted_indices[offset + i]] == 0){
      sorted_indices_out[offset + left_start] = sorted_indices[offset + i];
      sorted_labels_out[offset + left_start] = sorted_labels[offset + i];
      sorted_samples_out[offset + left_start] = sorted_samples[offset + i];
      left_start++;
    }
    else{
      sorted_indices_out[offset + right_start] = sorted_indices[offset + i];
      sorted_labels_out[offset + right_start] = sorted_labels[offset + i]; 
      sorted_samples_out[offset + right_start] = sorted_samples[offset + i];
      right_start++;
    }
  }
}






