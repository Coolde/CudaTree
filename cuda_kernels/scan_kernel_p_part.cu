#include "hip/hip_runtime.h"
#include<stdio.h>
#include<math.h>
#include<stdint.h>

#define MAX_NUM_SAMPLES %d
#define MAX_NUM_LABELS %d
#define THREADS_PER_BLOCK %d
#define LABEL_DATA_TYPE %s
#define COUNT_DATA_TYPE %s

__global__ void prefix_scan(LABEL_DATA_TYPE *sorted_targets, 
                        COUNT_DATA_TYPE *label_count,
                        int n_features, 
                        int n_samples, 
                        int stride){  
  int label_offset = blockIdx.x * MAX_NUM_LABELS * (blockDim.x + 1); //The offset of label_count for each block.
  int targets_offset = blockIdx.x * stride;                //The offset of sorted_targets for each block

  int range = ceil(double(n_samples) / blockDim.x);            //The range of elements each thread is reponsible for. 
  int n_active_threads = ceil(double(n_samples) / range);     //The number of threads that have the actual work to do.
  int range_begin =(threadIdx.x * range < n_samples)? threadIdx.x * range : n_samples;  //The begin index of each thread.
  int range_end = (range_begin + range < n_samples)? range_begin + range : n_samples;   //The end index of each thread

  //Initialize the first label_count of each thread
  for(int i = 0; i < MAX_NUM_LABELS; ++i)
    label_count[label_offset + i + threadIdx.x * MAX_NUM_LABELS] = 0;

  if(threadIdx.x == blockDim.x - 1)
    for(int i = 0; i < MAX_NUM_LABELS; ++i)
      label_count[label_offset + i + blockDim.x * MAX_NUM_LABELS] = 0;

  if(threadIdx.x < n_active_threads){  
    LABEL_DATA_TYPE cur_label = sorted_targets[targets_offset + range_begin];
    label_count[label_offset + cur_label + (threadIdx.x + 1) * MAX_NUM_LABELS]++;    
  }
 
  //Work out a range of label_count of each thread
  if(threadIdx.x < n_active_threads)
    for(int i = range_begin + 1; i < range_end; ++i){
      LABEL_DATA_TYPE cur_label = sorted_targets[targets_offset + i];
      label_count[label_offset + cur_label + (threadIdx.x + 1) * MAX_NUM_LABELS]++; 
    }
  
  __syncthreads();
  
  //Fist thread of the block does prefix sum on last element of label_count each thread
  if(threadIdx.x == 0)
    for(int i = 1; i < n_active_threads + 1; ++i)
    { 
      int last = (i - 1) * MAX_NUM_LABELS;
      int cur = i * MAX_NUM_LABELS;
      
      for(int l = 0; l < MAX_NUM_LABELS; ++l)
        label_count[label_offset + cur + l] += label_count[label_offset + last + l];    
    }
}
