#include "hip/hip_runtime.h"
#include<stdio.h>
#include<math.h>
#include<stdint.h>
#define IDX_DATA_TYPE %s
#define THREADS_PER_BLOCK %s

texture<char, 1> tex_mark;

__global__ void scan_reshuffle(
                          uint8_t* mark_table,
                          uint8_t* si_idx,
                          IDX_DATA_TYPE* sorted_indices_1,
                          IDX_DATA_TYPE* sorted_indices_2,
                          IDX_DATA_TYPE* begin_end_idx,
                          IDX_DATA_TYPE* split,
                          float *impurity,
                          uint16_t n_features,
                          uint32_t stride
                          ){  
  __shared__ IDX_DATA_TYPE last_sum;
  __shared__ IDX_DATA_TYPE shared_pos_table[THREADS_PER_BLOCK];
  IDX_DATA_TYPE reg_start_idx = begin_end_idx[2 * blockIdx.x];
  IDX_DATA_TYPE reg_stop_idx = begin_end_idx[2 * blockIdx.x + 1];
  IDX_DATA_TYPE reg_split_idx = split[blockIdx.x];
  IDX_DATA_TYPE n;
  
  if(reg_split_idx == reg_stop_idx)
    return;
  
  float imp_left = impurity[2 * blockIdx.x];
  float imp_right = impurity[2 * blockIdx.x + 1];

  if(imp_left == 0 && imp_right == 0)
    return;
  
  IDX_DATA_TYPE *p_sorted_indices_in;
  IDX_DATA_TYPE *p_sorted_indices_out;

  if(si_idx[blockIdx.x] == 0){
    p_sorted_indices_in = sorted_indices_1;
    p_sorted_indices_out = sorted_indices_2;
  }else{
    p_sorted_indices_in = sorted_indices_2;
    p_sorted_indices_out = sorted_indices_1;
  }
  
  if(imp_left != 0 || imp_right != 0)
    for(uint16_t shuffle_feature_idx = blockIdx.y; shuffle_feature_idx < n_features; shuffle_feature_idx += gridDim.y){
      uint32_t offset = shuffle_feature_idx * stride;

      if(threadIdx.x == 0)
        last_sum = 0;

      for(IDX_DATA_TYPE i = reg_start_idx; i < reg_stop_idx; i += blockDim.x){
        uint8_t side;
        IDX_DATA_TYPE idx = i + threadIdx.x;

        if(idx < reg_stop_idx)
          side = tex1Dfetch(tex_mark, p_sorted_indices_in[offset + idx]);
          //side = mark_table[p_sorted_indices_in[offset + idx]];
        
        shared_pos_table[threadIdx.x] = side;
        
        __syncthreads();

        for(uint16_t s = 1; s < blockDim.x; s *= 2){
          if(threadIdx.x >= s)
            n = shared_pos_table[threadIdx.x - s];
          else
            n = 0;
          __syncthreads();
          shared_pos_table[threadIdx.x] += n;
          __syncthreads();
        }
        
        IDX_DATA_TYPE reg_pos;
        
        if(i + threadIdx.x < reg_stop_idx){
          reg_pos = shared_pos_table[threadIdx.x] + last_sum;
          IDX_DATA_TYPE out_pos = (side == 1)? reg_start_idx + reg_pos - 1 : reg_split_idx + 1 + idx - reg_start_idx - reg_pos;
          p_sorted_indices_out[offset + out_pos] = p_sorted_indices_in[offset + idx];   
        }

        __syncthreads();

        if(threadIdx.x == blockDim.x - 1)
          last_sum = reg_pos;
      }

      __syncthreads();
    }
  else if(imp_left == 0)
    for(uint16_t shuffle_feature_idx = blockIdx.y; shuffle_feature_idx < n_features; shuffle_feature_idx += gridDim.y){
      uint32_t offset = shuffle_feature_idx * stride;

      if(threadIdx.x == 0)
        last_sum = 0;

      for(IDX_DATA_TYPE i = reg_start_idx; i < reg_stop_idx; i += blockDim.x){
        uint8_t side;
        IDX_DATA_TYPE idx = i + threadIdx.x;

        if(idx < reg_stop_idx)
          side = tex1Dfetch(tex_mark, p_sorted_indices_in[offset + idx]);
          //side = mark_table[p_sorted_indices_in[offset + idx]];
        
        shared_pos_table[threadIdx.x] = side;
        
        __syncthreads();

        for(uint16_t s = 1; s < blockDim.x; s *= 2){
          if(threadIdx.x >= s)
            n = shared_pos_table[threadIdx.x - s];
          else
            n = 0;
          __syncthreads();
          shared_pos_table[threadIdx.x] += n;
          __syncthreads();
        }
        
        IDX_DATA_TYPE reg_pos;
        reg_pos = shared_pos_table[threadIdx.x] + last_sum;
        
        if(side == 0 && i + threadIdx.x < reg_stop_idx){
          IDX_DATA_TYPE out_pos = reg_split_idx + 1 + idx - reg_start_idx - reg_pos;
          p_sorted_indices_out[offset + out_pos] = p_sorted_indices_in[offset + idx];   
        }

        __syncthreads();

        if(threadIdx.x == blockDim.x - 1)
          last_sum = reg_pos;
      }
      __syncthreads();
    }
  else
    for(uint16_t shuffle_feature_idx = blockIdx.y; shuffle_feature_idx < n_features; shuffle_feature_idx += gridDim.y){
      uint32_t offset = shuffle_feature_idx * stride;

      if(threadIdx.x == 0)
        last_sum = 0;

      for(IDX_DATA_TYPE i = reg_start_idx; i < reg_stop_idx; i += blockDim.x){
        uint8_t side;
        IDX_DATA_TYPE idx = i + threadIdx.x;

        if(idx < reg_stop_idx)
          side = tex1Dfetch(tex_mark, p_sorted_indices_in[offset + idx]);
          //side = mark_table[p_sorted_indices_in[offset + idx]];
        
        shared_pos_table[threadIdx.x] = side;
        
        __syncthreads();

        for(uint16_t s = 1; s < blockDim.x; s *= 2){
          if(threadIdx.x >= s)
            n = shared_pos_table[threadIdx.x - s];
          else
            n = 0;
          __syncthreads();
          shared_pos_table[threadIdx.x] += n;
          __syncthreads();
        }
        
        IDX_DATA_TYPE reg_pos;
        reg_pos = shared_pos_table[threadIdx.x] + last_sum;
        
        if(side == 1 && i + threadIdx.x < reg_stop_idx){
          IDX_DATA_TYPE out_pos =reg_start_idx + reg_pos - 1;
          p_sorted_indices_out[offset + out_pos] = p_sorted_indices_in[offset + idx];   
        }

        __syncthreads();

        if(threadIdx.x == blockDim.x - 1)
          last_sum = reg_pos;
      }
      __syncthreads();
    }
}

