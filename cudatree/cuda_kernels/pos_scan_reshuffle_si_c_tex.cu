#include "hip/hip_runtime.h"
#include<stdio.h>
#include<math.h>
#include<stdint.h>
#define IDX_DATA_TYPE %s
#define THREADS_PER_BLOCK %s

texture<char, 1> tex_mark;

__global__ void scan_reshuffle(uint8_t* mark_table,
                          IDX_DATA_TYPE* sorted_indices,
                          IDX_DATA_TYPE* sorted_indices_out,
                          int n_samples,
                          int split_idx,
                          int stride
                          ){  
  uint32_t indices_offset = blockIdx.x * stride;
  IDX_DATA_TYPE reg_pos = 0;
  uint32_t out_pos;
  uint32_t right_pos = indices_offset + split_idx + 1;
  uint8_t side;
  int n;

  __shared__ IDX_DATA_TYPE last_sum;
  __shared__ IDX_DATA_TYPE shared_pos_table[THREADS_PER_BLOCK];
  

  if(threadIdx.x == 0)
    last_sum = 0;
  
  for(IDX_DATA_TYPE i = threadIdx.x; i < n_samples; i += blockDim.x){
    side = tex1Dfetch(tex_mark, sorted_indices[indices_offset + i]);//mark_table[sorted_indices[indices_offset + i]];
    reg_pos = side;
    
    shared_pos_table[threadIdx.x] = reg_pos;

    __syncthreads();
     
    for(uint16_t s = 1; s < blockDim.x; s *= 2){
      if(threadIdx.x >= s){
        n = shared_pos_table[threadIdx.x - s];
      }
      else 
        n = 0;

      __syncthreads();
      shared_pos_table[threadIdx.x] += n;

      __syncthreads();
    }


    __syncthreads();
    
    reg_pos = shared_pos_table[threadIdx.x] + last_sum; 
    out_pos = (side == 1)? indices_offset + reg_pos - 1 : right_pos + i - reg_pos ;
    sorted_indices_out[out_pos] = sorted_indices[indices_offset + i];  
    
    __syncthreads();
    
    if(threadIdx.x == blockDim.x - 1)
      last_sum = reg_pos; 
  }
}

