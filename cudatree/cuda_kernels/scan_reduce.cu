#include "hip/hip_runtime.h"
#include<stdio.h>
#include<math.h>
#include<stdint.h>

#define MAX_NUM_LABELS %d
#define COUNT_DATA_TYPE %s
#define MAX_BLOCK_PER_FEATURE %d

__global__ void scan_reduce(
                        COUNT_DATA_TYPE *label_total_2d,
                        int n_block
                        ){

  uint32_t offset = blockIdx.x * (MAX_BLOCK_PER_FEATURE + 1) * MAX_NUM_LABELS;
  
  for(uint16_t i = 2; i <= n_block; ++i){
    
    uint32_t last_off = (i - 1) * MAX_NUM_LABELS;
    uint32_t this_off = i * MAX_NUM_LABELS;

    for(uint16_t t = threadIdx.x; t < MAX_NUM_LABELS; t += blockDim.x)
      label_total_2d[offset + this_off + t] += label_total_2d[offset + last_off + t];
  } 
}
